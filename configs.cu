
#include <hip/hip_runtime.h>
struct KernelConfig
{
    const int BlockRowWarps;
    const int BlockColWarps;
    const int WarpRowTiles;
    const int WarpColTiles;
    const int ChunkK;
    const int NumStages;
    const int PipelineStrategy;
    const int K;
    const int N;
};

constexpr KernelConfig octomul_4096_57344_8192 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 2,
    /* WarpRowTiles */ 3,
    /* WarpColTiles */ 4,
    /* ChunkK */ 4,
    /* NumStages */ 3,
    /* PipelineStrategy */ 1,
    /* K */ 8192,
    /* N */ 57344};

constexpr KernelConfig octomul_4096_8192_8192 = {
    /* BlockRowWarps */ 3,
    /* BlockColWarps */ 2,
    /* WarpRowTiles */ 3,
    /* WarpColTiles */ 4,
    /* ChunkK */ 4,
    /* NumStages */ 2,
    /* PipelineStrategy */ 3,
    /* K */ 8192,
    /* N */ 8192};

constexpr KernelConfig octomul_4096_28672_4096 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 2,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 4,
    /* ChunkK */ 4,
    /* NumStages */ 3,
    /* PipelineStrategy */ 1,
    /* K */ 4096,
    /* N */ 28672};

constexpr KernelConfig octomul_4096_10240_8192 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 3,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 4,
    /* ChunkK */ 4,
    /* NumStages */ 2,
    /* PipelineStrategy */ 1,
    /* K */ 8192,
    /* N */ 10240};

constexpr KernelConfig octomul_4096_6144_4096 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 3,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 3,
    /* ChunkK */ 4,
    /* NumStages */ 2,
    /* PipelineStrategy */ 3,
    /* K */ 4096,
    /* N */ 6144};

constexpr KernelConfig octomul_4096_4096_4096 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 2,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 4,
    /* ChunkK */ 4,
    /* NumStages */ 3,
    /* PipelineStrategy */ 1,
    /* K */ 4096,
    /* N */ 4096};

constexpr KernelConfig octomul_2048_8192_28672 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 3,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 4,
    /* ChunkK */ 2,
    /* NumStages */ 2,
    /* PipelineStrategy */ 3,
    /* K */ 28672,
    /* N */ 8192};

constexpr KernelConfig octomul_2048_10240_8192 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 3,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 4,
    /* ChunkK */ 2,
    /* NumStages */ 2,
    /* PipelineStrategy */ 3,
    /* K */ 8192,
    /* N */ 10240};

constexpr KernelConfig octomul_2048_8192_8192 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 2,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 4,
    /* ChunkK */ 2,
    /* NumStages */ 2,
    /* PipelineStrategy */ 3,
    /* K */ 8192,
    /* N */ 8192};

constexpr KernelConfig octomul_2048_28672_4096 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 3,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 4,
    /* ChunkK */ 2,
    /* NumStages */ 3,
    /* PipelineStrategy */ 3,
    /* K */ 4096,
    /* N */ 28672};

constexpr KernelConfig octomul_2048_6144_4096 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 2,
    /* WarpRowTiles */ 3,
    /* WarpColTiles */ 3,
    /* ChunkK */ 2,
    /* NumStages */ 3,
    /* PipelineStrategy */ 3,
    /* K */ 4096,
    /* N */ 6144};

constexpr KernelConfig octomul_2048_4096_4096 = {
    /* BlockRowWarps */ 3,
    /* BlockColWarps */ 2,
    /* WarpRowTiles */ 3,
    /* WarpColTiles */ 3,
    /* ChunkK */ 2,
    /* NumStages */ 3,
    /* PipelineStrategy */ 2,
    /* K */ 4096,
    /* N */ 4096};

constexpr KernelConfig octomul_1024_8192_28672 = {
    /* BlockRowWarps */ 4,
    /* BlockColWarps */ 3,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 3,
    /* ChunkK */ 2,
    /* NumStages */ 3,
    /* PipelineStrategy */ 2,
    /* K */ 28672,
    /* N */ 8192};

constexpr KernelConfig octomul_1024_6144_4096 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 2,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 3,
    /* ChunkK */ 2,
    /* NumStages */ 3,
    /* PipelineStrategy */ 2,
    /* K */ 4096,
    /* N */ 6144};

constexpr KernelConfig octomul_4096_4096_14336 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 3,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 4,
    /* ChunkK */ 4,
    /* NumStages */ 2,
    /* PipelineStrategy */ 1,
    /* K */ 14336,
    /* N */ 4096};

constexpr KernelConfig octomul_2048_57344_8192 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 3,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 4,
    /* ChunkK */ 2,
    /* NumStages */ 2,
    /* PipelineStrategy */ 2,
    /* K */ 8192,
    /* N */ 57344};

constexpr KernelConfig octomul_1024_57344_8192 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 4,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 3,
    /* ChunkK */ 4,
    /* NumStages */ 2,
    /* PipelineStrategy */ 3,
    /* K */ 8192,
    /* N */ 57344};

constexpr KernelConfig octomul_512_6144_4096 = {
    /* BlockRowWarps */ 4,
    /* BlockColWarps */ 2,
    /* WarpRowTiles */ 3,
    /* WarpColTiles */ 4,
    /* ChunkK */ 4,
    /* NumStages */ 2,
    /* PipelineStrategy */ 3,
    /* K */ 4096,
    /* N */ 6144};

constexpr KernelConfig octomul_1024_4096_14336 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 3,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 3,
    /* ChunkK */ 4,
    /* NumStages */ 2,
    /* PipelineStrategy */ 3,
    /* K */ 14336,
    /* N */ 4096};

constexpr KernelConfig octomul_1024_28672_4096 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 2,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 4,
    /* ChunkK */ 2,
    /* NumStages */ 3,
    /* PipelineStrategy */ 3,
    /* K */ 4096,
    /* N */ 28672};

constexpr KernelConfig octomul_2048_4096_14336 = {
    /* BlockRowWarps */ 3,
    /* BlockColWarps */ 2,
    /* WarpRowTiles */ 3,
    /* WarpColTiles */ 4,
    /* ChunkK */ 2,
    /* NumStages */ 3,
    /* PipelineStrategy */ 3,
    /* K */ 14336,
    /* N */ 4096};

constexpr KernelConfig octomul_512_57344_8192 = {
    /* BlockRowWarps */ 3,
    /* BlockColWarps */ 2,
    /* WarpRowTiles */ 3,
    /* WarpColTiles */ 4,
    /* ChunkK */ 4,
    /* NumStages */ 2,
    /* PipelineStrategy */ 1,
    /* K */ 8192,
    /* N */ 57344};

constexpr KernelConfig octomul_512_8192_28672 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 2,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 4,
    /* ChunkK */ 4,
    /* NumStages */ 3,
    /* PipelineStrategy */ 2,
    /* K */ 28672,
    /* N */ 8192};

constexpr KernelConfig octomul_512_4096_4096 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 3,
    /* WarpRowTiles */ 3,
    /* WarpColTiles */ 4,
    /* ChunkK */ 4,
    /* NumStages */ 2,
    /* PipelineStrategy */ 1,
    /* K */ 4096,
    /* N */ 4096};

constexpr KernelConfig octomul_512_28672_4096 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 3,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 3,
    /* ChunkK */ 4,
    /* NumStages */ 2,
    /* PipelineStrategy */ 1,
    /* K */ 4096,
    /* N */ 28672};

constexpr KernelConfig octomul_4096_8192_28672 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 3,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 4,
    /* ChunkK */ 4,
    /* NumStages */ 2,
    /* PipelineStrategy */ 2,
    /* K */ 28672,
    /* N */ 8192};

constexpr KernelConfig octomul_512_4096_14336 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 2,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 4,
    /* ChunkK */ 4,
    /* NumStages */ 3,
    /* PipelineStrategy */ 1,
    /* K */ 14336,
    /* N */ 4096};

constexpr KernelConfig octomul_512_8192_8192 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 2,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 4,
    /* ChunkK */ 4,
    /* NumStages */ 2,
    /* PipelineStrategy */ 1,
    /* K */ 8192,
    /* N */ 8192};

constexpr KernelConfig octomul_512_10240_8192 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 3,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 3,
    /* ChunkK */ 4,
    /* NumStages */ 2,
    /* PipelineStrategy */ 2,
    /* K */ 8192,
    /* N */ 10240};

constexpr KernelConfig octomul_1024_10240_8192 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 2,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 4,
    /* ChunkK */ 4,
    /* NumStages */ 2,
    /* PipelineStrategy */ 1,
    /* K */ 8192,
    /* N */ 10240};

constexpr KernelConfig octomul_1024_4096_4096 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 2,
    /* WarpRowTiles */ 4,
    /* WarpColTiles */ 3,
    /* ChunkK */ 4,
    /* NumStages */ 2,
    /* PipelineStrategy */ 1,
    /* K */ 4096,
    /* N */ 4096};

constexpr KernelConfig octomul_1024_8192_8192 = {
    /* BlockRowWarps */ 2,
    /* BlockColWarps */ 2,
    /* WarpRowTiles */ 3,
    /* WarpColTiles */ 4,
    /* ChunkK */ 4,
    /* NumStages */ 2,
    /* PipelineStrategy */ 1,
    /* K */ 8192,
    /* N */ 8192};
